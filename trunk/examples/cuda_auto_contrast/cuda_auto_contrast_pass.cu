#include "hip/hip_runtime.h"
/* Framework for Live Image Transformation (FLITr) 
 * Copyright (c) 2010 CSIR
 * 
 * This file is part of FLITr.
 *
 * FLITr is free software: you can redistribute it and/or modify it
 * under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of the
 * License, or (at your option) any later version.
 * 
 * FLITr is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * Lesser General Public License for more details.
 * 
 * You should have received a copy of the GNU Lesser General Public
 * License along with FLITr. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#ifndef CUDA_AUTO_CONTRAST_CU
#define CUDA_AUTO_CONTRAST_CU 1

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/binary_search.h>

__global__ 
void kernel_histeq(uchar4* trg, 
                   unsigned int imageWidth, unsigned int imageHeight, float scale, 
                   int* histTbl, 
                   unsigned char* src)
{
    // compute thread dimension
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if ( x < imageWidth && y < imageHeight )
    {
        unsigned int idx = x + y*imageWidth;
        unsigned char in_pix = src[idx];
        unsigned char out_pix = 255 * ((float)(histTbl[in_pix]) / scale);
        
        trg[idx] = make_uchar4(out_pix, out_pix, out_pix, 1);
    }
}

/// Functor to do contrast stretch
struct contrast_functor
{
    const int min;
    const float scale;

    contrast_functor(int im_min, int im_scale) : 
        min(im_min),
        scale(im_scale) {}

    __host__ __device__
    uchar4 operator()(const unsigned char& pix) const {
        unsigned char new_pix = ((float)(pix - min) / (float)scale) * 255;
        return make_uchar4(new_pix, new_pix, new_pix, 1);
    }
};

///////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS /////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////
extern "C"
void cu_histeq(const dim3& blocks, const dim3& threads, 
               void* trgBuffer, void* srcBuffer,
               unsigned int imageWidth, unsigned int imageHeight)
{
    thrust::device_ptr<unsigned char> d_data( reinterpret_cast<unsigned char*>(srcBuffer) );
    int sz = imageWidth*imageHeight;
    //thrust::sort(d_data, d_data+sz);
 
    // A copy is needed to in-place sort, because we still need the unsorted later
    thrust::device_vector<unsigned char> d_dataCpy(sz);
    thrust::copy(d_data, d_data+sz, d_dataCpy.begin());

    thrust::sort(d_dataCpy.begin(), d_dataCpy.end() );

    const int num_bins = 256;
    thrust::device_vector<int> d_cumulative_histogram(num_bins);
    
    // find the end of each bin of values
    thrust::counting_iterator<int> search_begin(0);
    thrust::upper_bound(d_dataCpy.begin(),
                        d_dataCpy.end(),
                        search_begin,
                        search_begin + num_bins,
                        d_cumulative_histogram.begin());
  
    int *d_ptr = thrust::raw_pointer_cast(&d_cumulative_histogram[0]);
    kernel_histeq<<< blocks, threads >>>( 
        reinterpret_cast<uchar4*>(trgBuffer), 
        imageWidth, imageHeight, imageWidth*imageHeight, 
        d_ptr, 
        reinterpret_cast<unsigned char*>(srcBuffer));
}

extern "C"
void cu_contrast_stretch(const dim3& blocks, const dim3& threads, 
                         void* trgBuffer, void* srcBuffer,
                         unsigned int imageWidth, unsigned int imageHeight)
{
    thrust::device_ptr<unsigned char> s_data( reinterpret_cast<unsigned char*>(srcBuffer) );
    thrust::device_ptr<uchar4> t_data( reinterpret_cast<uchar4*>(trgBuffer) );

    int sz = imageWidth*imageHeight;

    thrust::pair<thrust::device_ptr<unsigned char>, thrust::device_ptr<unsigned char> > minmax = thrust::minmax_element(s_data, s_data + sz);
    unsigned char immin = *minmax.first;
    unsigned char immax = *minmax.second;
    int scale = immax - immin;
    
    contrast_functor f(immin, scale);

    thrust::transform(s_data, s_data+sz, t_data, f);
}


#endif // CUDA_AUTO_CONTRAST_CU
