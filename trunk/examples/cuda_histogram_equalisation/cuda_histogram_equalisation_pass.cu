#include "hip/hip_runtime.h"
#ifndef CUDA_HISTOGRAM_EQUALIZATION_CU
#define CUDA_HISTOGRAM_EQUALIZATION_CU 1

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/binary_search.h>

__global__ 
void kernel_histeq(uchar4* trg, 
                   unsigned int imageWidth, unsigned int imageHeight, float scale, 
                   int* histTbl, 
                   unsigned char* src)
{
    // compute thread dimension
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if ( x < imageWidth && y < imageHeight )
    {
        unsigned int idx = x + y*imageWidth;
        unsigned char in_pix = src[idx];
        unsigned char out_pix = 255 * ((float)(histTbl[in_pix]) / scale);
        
        trg[idx] = make_uchar4(out_pix, out_pix, out_pix, 1);
    }
}

///////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS /////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////
extern "C"
void cu_histeq(const dim3& blocks, const dim3& threads, 
               void* trgBuffer, void* srcArray,
               unsigned int imageWidth, unsigned int imageHeight)
{
    thrust::device_ptr<unsigned char> d_data( reinterpret_cast<unsigned char*>(srcArray) );
    int sz = imageWidth*imageHeight;
    //thrust::sort(d_data, d_data+sz);
 
    // A copy is needed to in-place sort, because we still need the unsorted later
    thrust::device_vector<unsigned char> d_dataCpy(sz);
    thrust::copy(d_data, d_data+sz, d_dataCpy.begin());

    thrust::sort(d_dataCpy.begin(), d_dataCpy.end() );

    const int num_bins = 256;
    thrust::device_vector<int> d_cumulative_histogram(num_bins);
    
    // find the end of each bin of values
    thrust::counting_iterator<int> search_begin(0);
    thrust::upper_bound(d_dataCpy.begin(),
                        d_dataCpy.end(),
                        search_begin,
                        search_begin + num_bins,
                        d_cumulative_histogram.begin());
  
    int *d_ptr = thrust::raw_pointer_cast(&d_cumulative_histogram[0]);
    kernel_histeq<<< blocks, threads >>>( 
        reinterpret_cast<uchar4*>(trgBuffer), 
        imageWidth, imageHeight, imageWidth*imageHeight, 
        d_ptr, 
        reinterpret_cast<unsigned char*>(srcArray));
}

#endif // CUDA_HISTOGRAM_EQUALIZATION_CU
