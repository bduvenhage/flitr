#include "hip/hip_runtime.h"
/* Framework for Live Image Transformation (FLITr) 
 * Copyright (c) 2010 CSIR
 * 
 * This file is part of FLITr.
 *
 * FLITr is free software: you can redistribute it and/or modify it
 * under the terms of the GNU Lesser General Public License as
 * published by the Free Software Foundation, either version 3 of the
 * License, or (at your option) any later version.
 * 
 * FLITr is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * Lesser General Public License for more details.
 * 
 * You should have received a copy of the GNU Lesser General Public
 * License along with FLITr. If not, see
 * <http://www.gnu.org/licenses/>.
 */

#ifndef CUDA_AUTO_CONTRAST_CU
#define CUDA_AUTO_CONTRAST_CU 1

#include "pitched_2d_range.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/binary_search.h>

__global__ 
void kernel_histeq(uchar4* trg, int trg_pitch,
                   unsigned char* src, int src_pitch,
                   unsigned int imageWidth, unsigned int imageHeight, 
                   float scale, 
                   int* histTbl)
{
    // compute thread dimension
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if ( x < imageWidth && y < imageHeight )
    {
        unsigned int src_idx = x + (y * (src_pitch / sizeof(unsigned char)));
        unsigned int trg_idx = x + (y * (trg_pitch / sizeof(uchar4)));

        unsigned char in_pix = src[src_idx];
        unsigned char out_pix = 255 * ((float)(histTbl[in_pix]) / scale);
        
        trg[trg_idx] = make_uchar4(out_pix, out_pix, out_pix, 1);
    }
}

/// From count determine if we are in 2d bounds
struct in_2d_bound
{
    const int width;
    const int padded_width;
    const bool no_padding;

    in_2d_bound(int w, int pw) :
        width(w), padded_width(pw), no_padding(width == padded_width) {}

    __host__ __device__
    bool operator()(const int x)
    {
        if (no_padding) return true;

        return (x % padded_width) < width;
    }
};

/// Functor to do contrast stretch
struct contrast_functor
{
    const int min;
    const float scale;

    contrast_functor(int im_min, int im_scale) : 
        min(im_min),
        scale(im_scale) {}

    __host__ __device__
    uchar4 operator()(const unsigned char& pix) const {
        unsigned char new_pix = ((float)(pix - min) / (float)scale) * 255;
        return make_uchar4(new_pix, new_pix, new_pix, 1);
    }
};

///////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS /////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////
extern "C"
void cu_histeq(const dim3& blocks, const dim3& threads, 
               void* trgBuffer, int trgPitch, 
               void* srcBuffer, int srcPitch,
               unsigned int imageWidth, unsigned int imageHeight)
{
    thrust::device_ptr<unsigned char> src_data( reinterpret_cast<unsigned char*>(srcBuffer) );
    
    int src_padded_size = srcPitch/sizeof(unsigned char) * imageHeight;
    int src_size = imageWidth * imageHeight;
    
    //thrust::sort(src_data, src_data+src_padded_size);
    // A copy is needed to in-place sort, because we still need the unsorted later
    // we must cater for padded 2d input data, but pack it tightly for sort
    thrust::device_vector<unsigned char> src_data_copy(src_size);
    in_2d_bound copy_pred(imageWidth, srcPitch/sizeof(unsigned char));
    thrust::copy_if(src_data, src_data+src_padded_size, 
                    thrust::counting_iterator<int>(0),
                    src_data_copy.begin(),
                    copy_pred);
    
    thrust::sort(src_data_copy.begin(), src_data_copy.end());

    const int num_bins = 256;
    thrust::device_vector<int> d_cumulative_histogram(num_bins);
    
    // find the end of each bin of values
    thrust::counting_iterator<int> search_begin(0);
    thrust::upper_bound(src_data_copy.begin(),
                        src_data_copy.end(),
                        search_begin,
                        search_begin + num_bins,
                        d_cumulative_histogram.begin());
  
    int* histogram_ptr = thrust::raw_pointer_cast(&d_cumulative_histogram[0]);
    kernel_histeq<<< blocks, threads >>>( 
        reinterpret_cast<uchar4*>(trgBuffer), trgPitch,
        reinterpret_cast<unsigned char*>(srcBuffer), srcPitch,
        imageWidth, imageHeight, 
        imageWidth*imageHeight, 
        histogram_ptr
        );
}

extern "C"
void cu_contrast_stretch(const dim3& blocks, const dim3& threads, 
                         void* trgBuffer, int trgPitch, 
                         void* srcBuffer, int srcPitch,
                         unsigned int imageWidth, unsigned int imageHeight)
{
    thrust::device_ptr<unsigned char> s_data( reinterpret_cast<unsigned char*>(srcBuffer) );
    thrust::device_ptr<uchar4> t_data( reinterpret_cast<uchar4*>(trgBuffer) );

    int src_padded_size = srcPitch/sizeof(unsigned char) * imageHeight;
    int trg_padded_size = trgPitch/sizeof(uchar4) * imageHeight;

    // create iterators to skip padded elements
    typedef thrust::device_vector<unsigned char>::iterator src_it_type;
    pitched_2d_range<src_it_type> 
        src_it(s_data, s_data + src_padded_size, 
               imageWidth, srcPitch/sizeof(unsigned char));

    typedef thrust::device_vector<uchar4>::iterator trg_it_type;
    pitched_2d_range<trg_it_type> 
        trg_it(t_data, t_data + trg_padded_size, imageWidth, trgPitch/sizeof(uchar4));

    thrust::pair<pitched_2d_range<src_it_type>::iterator, 
                 pitched_2d_range<src_it_type>::iterator> minmax = thrust::minmax_element(src_it.begin(), src_it.end());
    unsigned char immin = *minmax.first;
    unsigned char immax = *minmax.second;
    int scale = immax - immin;
    
    contrast_functor f(immin, scale);

    thrust::transform(src_it.begin(), src_it.end(), trg_it.begin(), f);
}


#endif // CUDA_AUTO_CONTRAST_CU
